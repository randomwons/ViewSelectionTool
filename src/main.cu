#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include "loader/depth.h"
#include "loader/image.h"
#include "loader/intrinsic.h"
#include "loader/pose.h"

struct Ray {
    double3 o, d;
    __device__ __host__ Ray(double3 o, double3 d) : o(o), d(d) {}
};

class Octree {
public:
    float3 min, max, center;
    float resolution;

    __device__ __host__ Octree(float3 min, float3 max, float resolution) 
        : min(min), max(max), resolution(resolution) {
            center = make_float3((max.x - min.x) / 2, (max.y - min.y) / 2, (max.z - min.z) / 2); 
         }

    __device__ bool traverse(const Ray& ray) {
        
        // double3 center = make_double3(0, 0, 0);
        // double radius = 0.1;

        // double3 oc = make_double3(ray.o.x - center.x, ray.o.y - center.y, ray.o.z - center.z);
        // double a = ray.d.x * ray.d.x + ray.d.y * ray.d.y + ray.d.z * ray.d.z;
        // double b = 2. * (oc.x * ray.d.x + oc.y * ray.d.y + oc.z * ray.d.z);
        // double c = oc.x * oc.x + oc.y * oc.y + oc.z * oc.z - radius * radius;

        // double dis = b*b - 4*a*c;
        // return (dis >= 0);

        double rayorix;
        double rayoriy;
        double rayoriz;

        double rayInvDirx;
        double rayInvDiry;
        double rayInvDirz;

        if(ray.d.x < 0.0f){
            rayorix = center.x * 2.0f - ray.o.x;
            rayInvDirx = -(1 / ray.d.x);
        } else {
            rayorix = ray.o.x;
            rayInvDirx = 1 / ray.d.x; 
        }
        if(ray.d.y < 0.0f){
            rayoriy = center.y * 2.0f - ray.o.y;
            rayInvDiry = -(1 / ray.d.y);
        } else {
            rayoriy = ray.o.y;
            rayInvDiry = 1 / ray.d.y; 
        }
        if(ray.d.z < 0.0f){
            rayoriz = center.z * 2.0f - ray.o.z;
            rayInvDirz = -(1 / ray.d.z);
        } else {
            rayoriz = ray.o.z;
            rayInvDirz = 1 / ray.d.z; 
        }

        const float tx0 = (min.x - rayorix) * rayInvDirx;
        const float tx1 = (max.x - rayorix) * rayInvDirx;
        const float ty0 = (min.y - rayoriy) * rayInvDiry;
        const float ty1 = (max.y - rayoriy) * rayInvDiry;
        const float tz0 = (min.z - rayoriz) * rayInvDirz;
        const float tz1 = (max.z - rayoriz) * rayInvDirz;

        if(fmaxf(fmaxf(tx0, ty0), tz0) < fminf(fminf(tx1, ty1), tz1)) return true;
        return false;
       
    }

};

__global__ void generateRays(Ray* rays, int width, int height, double* intrinsic, double* pose){

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= width || y >= height) return;

    double fx = intrinsic[0];
    double fy = intrinsic[4];
    double cx = intrinsic[2];
    double cy = intrinsic[5];

    double dx = (x - cx) / fx;
    double dy = (y - cy) / fy;
    double dz = 1;

    double worlddx = pose[0 * 4 + 0] * dx + pose[1 * 4 + 0] * dy + pose[2 * 4 + 0] * dz;
    double worlddy = pose[0 * 4 + 1] * dx + pose[1 * 4 + 1] * dy + pose[2 * 4 + 1] * dz;
    double worlddz = pose[0 * 4 + 2] * dx + pose[1 * 4 + 2] * dy + pose[2 * 4 + 2] * dz;

    double norm = norm3d(worlddx, worlddy, worlddz);
    worlddx /= norm;
    worlddy /= norm;
    worlddz /= norm;

    double ox = -(pose[0 * 4 + 0] * pose[0 * 4 + 3] + pose[1 * 4 + 0] * pose[1 * 4 + 3] + pose[2 * 4 + 0] * pose[2 * 4 + 3]);
    double oy = -(pose[0 * 4 + 1] * pose[0 * 4 + 3] + pose[1 * 4 + 1] * pose[1 * 4 + 3] + pose[2 * 4 + 1] * pose[2 * 4 + 3]);
    double oz = -(pose[0 * 4 + 2] * pose[0 * 4 + 3] + pose[1 * 4 + 2] * pose[1 * 4 + 3] + pose[2 * 4 + 2] * pose[2 * 4 + 3]);

    int pid = y * width + x;
    rays[pid].o = make_double3(ox, oy, oz);
    rays[pid].d = make_double3(worlddx, worlddy, worlddz);

}

__global__ void raytracing(uchar4* data, Octree* octree, Ray* rays, int width, int height) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= width || y >= height) return;

    int pid = y * width + x;
    // if(x >= width / 2 - 10 && x <= width /2 + 10 &&
    //    y >= height / 2 - 10 && y <= height /2 + 10) {
    //     data[pid].x = 255;
    //     data[pid].y = 0;
    //     data[pid].z = 255;
    //     data[pid].w = 255;
    //     return;    
    // }


    if(octree->traverse(rays[pid])) {
        data[pid].x = 255;
        data[pid].y = 255;
        data[pid].z = 255;
        data[pid].w = 255;
        return;    
    }
    // printf("max : %f, %f, %f\n", octree->max.x, octree->max.y, octree->max.z);
    // data[pid].x = (unsigned char)value;
    data[pid].x = (unsigned char)(__saturatef(rays[pid].d.x) * 255.0f);
    data[pid].y = (unsigned char)(__saturatef(rays[pid].d.y) * 255.0f);
    data[pid].z = (unsigned char)(__saturatef(rays[pid].d.z) * 255.0f);
    data[pid].w = 255;
}

// __global__ void setOctree(Octree* octree) {
//     if(threadIdx.x != 0) return;

//     octree = new Octree(make_float3(0, 0, 0), make_float3(1.28, 1.28, 1.28), 0.01);
// }


int main() {

    Octree* octree = new Octree(make_float3(-0.5, -0.5, -0.5), make_float3(0.5, 0.5, 0.5), 0.01);
    Octree* d_octree;
    hipMalloc((void**)&d_octree, sizeof(Octree));
    hipMemcpy(d_octree, octree, sizeof(Octree), hipMemcpyHostToDevice);
    // setOctree<<<1, 1>>>(d_octree);

    // int N = 1;
    for(int N = 0; N < 160; N++){
    DepthNpy depth("C:/DATASET/dataset/armadillo/depth/" + std::to_string(N) + ".npy");
    Image image("C:/DATASET/dataset/armadillo/color/" + std::to_string(N) + ".png");
    Intrinsic intrinsic("C:/DATASET/dataset/armadillo/intrinsic/" + std::to_string(N) + ".txt");
    Pose pose("C:/DATASET/dataset/armadillo/pose/" + std::to_string(N) + ".txt");

    int width = depth.width();
    int height = depth.height();


    dim3 threadLayout(32, 32);
    dim3 gridLayout(width / 32 + 1, height / 32 + 1);

    Ray* rays;
    hipMalloc((void**)&rays, sizeof(Ray) * width * height);

    uchar4* data;
    hipMalloc((void**)&data, sizeof(uchar4) * width * height);

    double* d_intrinsic;
    hipMalloc((void**)&d_intrinsic, sizeof(double) * 9);
    hipMemcpy(d_intrinsic, intrinsic.data, sizeof(double) * 9, hipMemcpyHostToDevice);

    double* d_pose;
    hipMalloc((void**)&d_pose, sizeof(double) * 16);
    hipMemcpy(d_pose, pose.data, sizeof(double) * 16, hipMemcpyHostToDevice);

    generateRays<<<gridLayout, threadLayout>>>(rays, width, height, d_intrinsic, d_pose);
    raytracing<<<gridLayout, threadLayout>>>(data, d_octree, rays, width, height);
    hipDeviceSynchronize();

    uchar4* output = new uchar4[width * height];
    hipMemcpy(output, data, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);

    stbi_write_png(("data/outputtest" + std::to_string(N) + ".png").c_str(), width, height, 4, (void*)output, width * 4);

    delete[] output;
    hipFree(data);
    hipFree(rays);
    hipFree(d_pose);
    hipFree(d_intrinsic);
    }


    hipFree(d_octree);
    return 0;

}
