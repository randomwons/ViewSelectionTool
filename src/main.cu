#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include "loader/depth.h"
#include "loader/image.h"
#include "loader/intrinsic.h"
#include "loader/pose.h"

struct Ray {
    double3 o, d;
    __device__ __host__ Ray(double3 o, double3 d) : o(o), d(d) {}
};

class Octree {
public:
    float3 min, max, center;
    float resolution;

    __device__ __host__ Octree(float3 min, float3 max, float resolution) 
        : min(min), max(max), resolution(resolution) {
            center = make_float3((max.x - min.x) / 2, (max.y - min.y) / 2, (max.z - min.z) / 2); 
         }

    __device__ bool traverse(const Ray& ray) {
        
        double3 center = make_double3(0, 0, 0);
        double radius = 0.1;

        double3 oc = make_double3(ray.o.x - center.x, ray.o.y - center.y, ray.o.z - center.z);
        double a = ray.d.x * ray.d.x + ray.d.y * ray.d.y + ray.d.z * ray.d.z;
        double b = 2. * (oc.x * ray.d.x + oc.y * ray.d.y + oc.z * ray.d.z);
        double c = oc.x * oc.x + oc.y * oc.y + oc.z * oc.z - radius * radius;

        double dis = b*b - 4*a*c;
        return (dis >= 0);
       
    }

};

__global__ void generateRays(Ray* rays, int width, int height, double* intrinsic, double* pose){

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= width || y >= height) return;

    double fx = intrinsic[0];
    double fy = intrinsic[4];
    double cx = intrinsic[2];
    double cy = intrinsic[5];

    // double fx = 1000.;
    // double fy = 1000.;
    // double cx = 640.;
    // double cy = 360.;

    double dx = (x - cx) / fx;
    double dy = (y - cy) / fy;
    double dz = 1;

    // double norm = norm3d(dx, dy, dz);
    // dx /= norm;
    // dy /= norm;
    // dz /= norm;

    dx = pose[0 * 4 + 0] * dx + pose[1 * 4 + 0] * dy + pose[2 * 4 + 0] * dz;
    dy = pose[0 * 4 + 1] * dx + pose[1 * 4 + 1] * dy + pose[2 * 4 + 1] * dz;
    dz = pose[0 * 4 + 2] * dx + pose[1 * 4 + 2] * dy + pose[2 * 4 + 2] * dz;
    // dx = pose[0 * 4 + 0] * dx + pose[0 * 4 + 1] * dy + pose[0 * 4 + 2] * dz;
    // dy = pose[1 * 4 + 0] * dx + pose[1 * 4 + 1] * dy + pose[1 * 4 + 2] * dz;
    // dz = pose[2 * 4 + 0] * dx + pose[2 * 4 + 1] * dy + pose[2 * 4 + 2] * dz;
        double norm = norm3d(dx, dy, dz);
    dx /= norm;
    dy /= norm;
    dz /= norm;
    // double ox = pose[0 * 4 + 3];
    // double oy = pose[1 * 4 + 3];
    // double oz = pose[2 * 4 + 3];
    double ox = -(pose[0 * 4 + 0] * pose[0 * 4 + 3] + pose[1 * 4 + 0] * pose[1 * 4 + 3] + pose[2 * 4 + 0] * pose[2 * 4 + 3]);
    double oy = -(pose[0 * 4 + 1] * pose[0 * 4 + 3] + pose[1 * 4 + 1] * pose[1 * 4 + 3] + pose[2 * 4 + 1] * pose[2 * 4 + 3]);
    double oz = -(pose[0 * 4 + 2] * pose[0 * 4 + 3] + pose[1 * 4 + 2] * pose[1 * 4 + 3] + pose[2 * 4 + 2] * pose[2 * 4 + 3]);

    int pid = y * width + x;
    rays[pid].o = make_double3(ox, oy, oz);
    rays[pid].d = make_double3(dx, dy, dz);

    if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        for(int i = 0; i < 4; i++){
            printf("[CUDA] %f, %f, %f, %f\n", pose[i * 4 + 0], pose[i * 4 + 1], pose[i * 4 + 2], pose[i * 4 + 3]);
        }
        printf("[INV] %f, %f, %f, %f\n", pose[0 * 4 + 0], pose[1 * 4 + 0], pose[2 * 4 + 0], ox);
        printf("[INV] %f, %f, %f, %f\n", pose[0 * 4 + 1], pose[1 * 4 + 1], pose[2 * 4 + 1], oy);
        printf("[INV] %f, %f, %f, %f\n", pose[0 * 4 + 2], pose[1 * 4 + 2], pose[2 * 4 + 2], oz);
        printf("[INV] %f, %f, %f, %f\n", pose[3 * 4 + 0], pose[3 * 4 + 0], pose[3 * 4 + 0], 1.);
    }

}

__global__ void raytracing(uchar4* data, Octree* octree, Ray* rays, int width, int height) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= width || y >= height) return;

    int pid = y * width + x;
    if(x >= width / 2 - 10 && x <= width /2 + 10 &&
       y >= height / 2 - 10 && y <= height /2 + 10) {
        data[pid].x = 255;
        data[pid].y = 0;
        data[pid].z = 255;
        data[pid].w = 255;
        return;    
    }


    if(octree->traverse(rays[pid])) {
        data[pid].x = 255;
        data[pid].y = 255;
        data[pid].z = 255;
        data[pid].w = 255;
        return;    
    }
    // printf("max : %f, %f, %f\n", octree->max.x, octree->max.y, octree->max.z);
    // data[pid].x = (unsigned char)value;
    data[pid].x = (unsigned char)(__saturatef(rays[pid].d.x) * 255.0f);
    data[pid].y = (unsigned char)(__saturatef(rays[pid].d.y) * 255.0f);
    data[pid].z = (unsigned char)(__saturatef(rays[pid].d.z) * 255.0f);
    data[pid].w = 255;
}

// __global__ void setOctree(Octree* octree) {
//     if(threadIdx.x != 0) return;

//     octree = new Octree(make_float3(0, 0, 0), make_float3(1.28, 1.28, 1.28), 0.01);
// }


int main() {

    Octree* octree = new Octree(make_float3(0, 0, 0), make_float3(1.28, 1.28, 1.28), 0.01);
    Octree* d_octree;
    hipMalloc((void**)&d_octree, sizeof(Octree));
    hipMemcpy(d_octree, octree, sizeof(Octree), hipMemcpyHostToDevice);
    // setOctree<<<1, 1>>>(d_octree);

    // int N = 1;
    for(int N = 0; N < 1; N++){
    DepthNpy depth("C:/DATASET/dataset/armadillo/depth/" + std::to_string(N) + ".npy");
    Image image("C:/DATASET/dataset/armadillo/color/" + std::to_string(N) + ".png");
    Intrinsic intrinsic("C:/DATASET/dataset/armadillo/intrinsic/" + std::to_string(N) + ".txt");
    Pose pose("C:/DATASET/dataset/armadillo/pose/" + std::to_string(N) + ".txt");
    pose.print();
    int width = depth.width();
    int height = depth.height();
    // int width = 1280;
    // int height = 720;

    dim3 threadLayout(32, 32);
    dim3 gridLayout(width / 32 + 1, height / 32 + 1);

    Ray* rays;
    hipMalloc((void**)&rays, sizeof(Ray) * width * height);

    uchar4* data;
    hipMalloc((void**)&data, sizeof(uchar4) * width * height);

    double* d_intrinsic;
    hipMalloc((void**)&d_intrinsic, sizeof(double) * 9);
    hipMemcpy(d_intrinsic, intrinsic.data, sizeof(double) * 9, hipMemcpyHostToDevice);

    double* d_pose;
    // double t_pose[] = {
    //     1, 0,  0, 0.1,
    //     0, 1,  0, 0.0,
    //     0, 0,  1, 1,
    //     0, 0, 0, 1
    // };
    double t_pose[] = {
        0.0000000, -1.0000000,  0.0000000, 0.1,
        1.0000000,  0.0000000,  0.0000000, 0.0,
        0.0000000,  0.0000000,  1.0000000, 1,
        0, 0, 0, 1
    };
    hipMalloc((void**)&d_pose, sizeof(double) * 16);
    // hipMemcpy(d_pose, pose.data, sizeof(double) * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_pose, t_pose, sizeof(double) * 16, hipMemcpyHostToDevice);

    generateRays<<<gridLayout, threadLayout>>>(rays, width, height, d_intrinsic, d_pose);
    raytracing<<<gridLayout, threadLayout>>>(data, d_octree, rays, width, height);
    hipDeviceSynchronize();

    uchar4* output = new uchar4[width * height];
    hipMemcpy(output, data, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);

    stbi_write_png(("outputtest" + std::to_string(N) + ".png").c_str(), width, height, 4, (void*)output, width * 4);

    delete[] output;
    hipFree(data);
    hipFree(rays);
    hipFree(d_pose);
    hipFree(d_intrinsic);
    }


    hipFree(d_octree);
    return 0;

}
